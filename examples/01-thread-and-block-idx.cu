#include "hip/hip_runtime.h"
'''
@File    :   01-thread-and-block-idx.cu
@Time    :   2021/04/22 18:04:40
@Author  :   Haoyu Guo 
@Version :   1.1
@Contact :   haoyuguo@usc.edu
'''
#include<stdio.h>

// Currently the 01-thread-and-block-idx.cu file contains a working kernel 
// that is printing a failure message. Open the file to learn how to update 
// the execution configuration so that the success message will print. After refactoring, 
// compile and run the code with the code execution cell below to confirm your work. 
// Refer to the solution if you get stuck.

__global__ void printSuccessForCorrectExecutionConfiguration(){
    if(threadIdx.x == 9 && blockIdx.x == 9){
        printf("Success!\n");
    } 
    else {
        printf("Failure. Update the execution configuration as necessary.\n");
  }
}


int main()
{
  /*
   * Update the execution configuration so that the kernel
   * will print `"Success!"`.
   */

  printSuccessForCorrectExecutionConfiguration<<<10,10>>>();
  hipDeviceSynchronize();
  // printSuccessForCorrectExecutionConfiguration<<<1, 1>>>();
}


