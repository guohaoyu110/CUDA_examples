
#include <hip/hip_runtime.h>
#include <stdio.h>

// 执行的指令
// nvcc -arch=sm_70 -o first-parallel 01-first-parallel.cu -run

/*
 * Refactor firstParallel so that it can run on the GPU.
 */

//  void firstParallel(){
//     printf("This should be running in parallel. \n");
//  }
__global__ void firstParallel(){
    printf("This should be running in parallel. \n");
}

 int main(){
   /*
   * Refactor this call to firstParallel to execute in parallel
   * on the GPU.
   */
    firstParallel<<<5, 5>>>();
    // <<< NUMBER_OF_BLOCKS, NUMBER_OF_THREADS_PER_BLOCK>>>
    hipDeviceSynchronize();
    /*
   * Refactor this call to firstParallel to execute in parallel
   * on the GPU.
   */
 }

 // 这样就可以执行25次一共。