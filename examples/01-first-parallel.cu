#include "hip/hip_runtime.h"
'''
@File    :   01-first-parallel.cu
@Time    :   2021/04/22 18:05:36
@Author  :   Haoyu Guo 
@Version :   1.0
@Contact :   haoyuguo@usc.edu
'''
#include<stidio.h>

// 执行的指令
// nvcc -arch=sm_70 -o first-parallel 01-first-parallel.cu -run

/*
 * Refactor firstParallel so that it can run on the GPU.
 */

//  void firstParallel(){
//     printf("This should be running in parallel. \n");
//  }
__global__ void firstParallel(){
    printf("This should be running in parallel. \n");
}

 int main(){
   /*
   * Refactor this call to firstParallel to execute in parallel
   * on the GPU.
   */
    firstParallel<<<5, 5>>>();
    // <<< NUMBER_OF_BLOCKS, NUMBER_OF_THREADS_PER_BLOCK>>>
    hipDeviceSynchronize();
    /*
   * Refactor this call to firstParallel to execute in parallel
   * on the GPU.
   */
 }

 // 这样就可以执行25次一共。