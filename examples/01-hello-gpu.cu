#include "hip/hip_runtime.h"
'''
@File    :   01-hello-gpu.cu
@Time    :   2021/04/22 18:05:30
@Author  :   Haoyu Guo 
@Version :   1.0
@Contact :   haoyuguo@usc.edu
'''
#include<stidio.h>
// 执行下面的指令
// nvcc -arch=sm_70 -o hello-gpu 01-hello-gpu.cu -run
void helloCPU()
{
  printf("Hello from the CPU.\n");
}

/*
 * Refactor the `helloGPU` definition to be a kernel
 * that can be launched on the GPU. Update its message
 * to read "Hello from the GPU!"
 */

 __global__ void helloGPU() 
{
  printf("Hello also from the GPU.\n");
}

int main()
{

  helloCPU();

  /*
   * Refactor this call to `helloGPU` so that it launches
   * as a kernel on the GPU.
   */

  //helloGPU();
  helloGPU<<<1, 1>>>();

  /*
   * Add code below to synchronize on the completion of the
   * `helloGPU` kernel completion before continuing the CPU
   * thread.
   */
   hipDeviceSynchronize();
}
