
#include <hip/hip_runtime.h>
#include <stdio.h>


// nvcc -arch=sm_70 -o multi-block-loop 02-multi-block-loop.cu -run

/*
 * Refactor `loop` to be a CUDA Kernel. The new kernel should
 * only do the work of 1 iteration of the original loop.
 */

// void loop(int N)
// {
//   for (int i = 0; i < N; ++i)
//   {
//     printf("This is iteration number %d\n", i);
//   }
// }

__global__ void loop(){
    printf("This is iteration number %d\n", blockIdx.x * blockDim.x + threadIdx.x ) ;
}

int main()
{
  /*
   * When refactoring `loop` to launch as a kernel, be sure
   * to use the execution configuration to control how many
   * "iterations" to perform.
   *
   * For this exercise, be sure to use more than 1 block in
   * the execution configuration.
   */

  // int N = 10;
  // loop<<<1,10>>> ();
  loop<<<2,5>>> ();
  hipDeviceSynchronize();
  // loop(N);
}
